#include "hip/hip_runtime.h"
#include "Macro.h"
#include "CUFLU.h"

#if ( defined GPU  &&  MODEL == ELBDM )



// useful macros
#define to1D1(z,y,x) ( __umul24(z, FLU_NXT*FLU_NXT) + __umul24(y, FLU_NXT) + x )
#define to1D2(z,y,x) ( __umul24(z-FLU_GHOST_SIZE, PS2*PS2) + __umul24(y-FLU_GHOST_SIZE, PS2) + x-FLU_GHOST_SIZE )

#ifdef LAPLACIAN_4TH
#  define LAP_GHOST  2
#  define LAP1(In,i)    (  real(1.0/ 12.0)*( - In[i-2] + (real)16.0*In[i-1] - (real)30.0*In[i  ] \
                                             - In[i+2] + (real)16.0*In[i+1] )  )
#  define LAP2(In,i)    (  real(1.0/144.0)*( + In[i-4] - (real)32.0*In[i-3] + (real)316.0*In[i-2] - (real)992.0*In[i-1] \
                                             + In[i+4] - (real)32.0*In[i+3] + (real)316.0*In[i+2] - (real)992.0*In[i+1] \
                                             +  (real)1414.0*In[i  ] )  )
#  ifndef CONSERVE_MASS
#  define LAP3(In,i)    (  real(1.0/1728.0)* \
       (  -In[i-6] + (real)48*In[i-5] - (real)858*In[i-4] + (real)7024*In[i-3] - (real)27279*In[i-2] + (real)58464*In[i-1] \
          -In[i+6] + (real)48*In[i+5] - (real)858*In[i+4] + (real)7024*In[i+3] - (real)27279*In[i+2] + (real)58464*In[i+1] \
          - (real)74796*In[i  ] )  )
#  endif

#else // #ifdef LAPLACIAN_4TH

#  define LAP_GHOST     1
#  define LAP1(In,i)    ( + In[i-1] - (real)2.0*In[i  ] + In[i+1] )
#  define LAP2(In,i)    ( + In[i-2] - (real)4.0*In[i-1] + (real)6.0*In[i  ] - (real)4.0*In[i+1] + In[i+2] )
#  ifndef CONSERVE_MASS
#  define LAP3(In,i)    ( + In[i-3] - (real)6.0*In[i-2] + (real)15.0*In[i-1] - (real)20.0*In[i  ] \
                          + In[i+3] - (real)6.0*In[i+2] + (real)15.0*In[i+1] )

#  endif

#endif // #ifdef LAPLACIAN_4TH ... else ...


static __device__ void CUFLU_Advance( real g_Fluid_In [][FLU_NIN ][ FLU_NXT*FLU_NXT*FLU_NXT ],
                                      real g_Fluid_Out[][FLU_NOUT][ PS2*PS2*PS2 ],
                                      real g_Flux     [][9][NFLUX_TOTAL][ PS2*PS2 ],
                                      const real dt, const real _dh, const real Eta1, const real Eta2, const bool StoreFlux,
                                      const real Taylor3_Coeff, const uint j_gap, const uint k_gap,
                                      real s_In[][FLU_BLOCK_SIZE_Y][FLU_NXT], real s_Half[][FLU_BLOCK_SIZE_Y][FLU_NXT],
                                      real s_Flux1[][PS2+1], real s_Flux2[][PS2+1], const bool FinalOut, const int XYZ, const real MinDens );




//-------------------------------------------------------------------------------------------------------
// Function    :  CUFLU_ELBDMSolver
// Description :  GPU ELBDM kinematic solver based on expanding the propagator to 3rd order
//
// Note        :  1. The three-dimensional evolution is achieved by applying x, y, and z operators successively.
//                   Since these operators commute, the order of applying them are irrelevant.
//                   --> Input pamameter "XYZ" is actually useless
//                   --> Nevertheless, the symmetry in different directions will be broken if CONSERVE_MASS is on
//                2. The implementation is very similar to the function " CUFLU_FluidSolver_RTVD"
//                4. Prefix "g" for pointers pointing to the "Global" memory space
//                   Prefix "s" for pointers pointing to the "Shared" memory space
//
// Parameter   :  g_Fluid_In     : Global memory array storing the input variables
//                g_Fluid_Out    : Global memory array to store the output variables
//                g_Flux         : Global memory array to store the output fluxes (useful only if StoreFlux == true)
//                dt             : Time interval to advance solution
//                _dh            : 1 / grid size
//                Eta            : Particle mass / Planck constant
//                StoreFlux      : true --> store the coarse-fine fluxes
//                                      --> useful only if CONSERVE_MASS is defined
//                Taylor3_Coeff  : Coefficient in front of the third term in the Taylor expansion
//                XYZ            : true  : x->y->z ( forward sweep)
//                                 false : z->y->x (backward sweep)
//                                 --> Meaningless if CONSERVE_MASS is off since the operators along different directions
//                                     commute
//                                 --> Meaningful if CONSERVE_MASS is on, in which the symmetry along different directions
//                                     are broken ...
//                MinDens        : Minimum allowed density
//-------------------------------------------------------------------------------------------------------
__global__ void CUFLU_ELBDMSolver( real g_Fluid_In [][FLU_NIN ][ FLU_NXT*FLU_NXT*FLU_NXT ],
                                   real g_Fluid_Out[][FLU_NOUT][ PS2*PS2*PS2 ],
                                   real g_Flux     [][9][NFLUX_TOTAL][ PS2*PS2 ],
                                   const real dt, const real _dh, const real Eta1, const real Eta2, const bool StoreFlux,
                                   const real Taylor3_Coeff, const bool XYZ, const real MinDens )
{

   __shared__ real s_In  [FLU_NIN][FLU_BLOCK_SIZE_Y][FLU_NXT];
#  ifdef CONSERVE_MASS
   __shared__ real s_Half[FLU_NIN][FLU_BLOCK_SIZE_Y][FLU_NXT];
   __shared__ real s_Flux1[FLU_BLOCK_SIZE_Y][PS2+1];
   __shared__ real s_Flux2[FLU_BLOCK_SIZE_Y][PS2+1];
#  else
   real (*s_Half)[FLU_BLOCK_SIZE_Y][FLU_NXT] = NULL;  // useless if CONSERVE_MASS is off
   real (*s_Flux1)[PS2+1]                     = NULL;  // useless if CONSERVE_MASS is off
   real (*s_Flux2)[PS2+1]                     = NULL;  // useless if CONSERVE_MASS is off
#  endif

   if ( XYZ )
   {
      CUFLU_Advance( g_Fluid_In, g_Fluid_Out, g_Flux, dt, _dh, Eta1, Eta2, StoreFlux, Taylor3_Coeff,
                                  0,              0, s_In, s_Half, s_Flux1, s_Flux2, false, 0, MinDens );
      CUFLU_Advance( g_Fluid_In, g_Fluid_Out, g_Flux, dt, _dh, Eta1, Eta2, StoreFlux, Taylor3_Coeff,
                     FLU_GHOST_SIZE,              0, s_In, s_Half, s_Flux1, s_Flux2, false, 3, MinDens );
      CUFLU_Advance( g_Fluid_In, g_Fluid_Out, g_Flux, dt, _dh, Eta1, Eta2, StoreFlux, Taylor3_Coeff,
                     FLU_GHOST_SIZE, FLU_GHOST_SIZE, s_In, s_Half, s_Flux1, s_Flux2,  true, 6, MinDens );
   }

   else
   {
      CUFLU_Advance( g_Fluid_In, g_Fluid_Out, g_Flux, dt, _dh, Eta1, Eta2, StoreFlux, Taylor3_Coeff,
                                  0,              0, s_In, s_Half, s_Flux1, s_Flux2, false, 6, MinDens );
      CUFLU_Advance( g_Fluid_In, g_Fluid_Out, g_Flux, dt, _dh, Eta1, Eta2, StoreFlux, Taylor3_Coeff,
                                  0, FLU_GHOST_SIZE, s_In, s_Half, s_Flux1, s_Flux2, false, 3, MinDens );
      CUFLU_Advance( g_Fluid_In, g_Fluid_Out, g_Flux, dt, _dh, Eta1, Eta2, StoreFlux, Taylor3_Coeff,
                     FLU_GHOST_SIZE, FLU_GHOST_SIZE, s_In, s_Half, s_Flux1, s_Flux2,  true, 0, MinDens );
   }

} // FUNCTION : CUFLU_ELBDMSolver



//-------------------------------------------------------------------------------------------------------
// Function    :  CUFLU_Advance
// Description :  Use GPU to advance solutions by one time-step
//
// Note        :  1. Based on expanding the kinematic propagator to 3rd order
//                2. Prefix "g" for pointers pointing to the "Global" memory space
//                   Prefix "s" for pointers pointing to the "Shared" memory space
//                3. The direction of the one dimensional sweep is determined by the input parameter "XYZ"
//
// Parameter   :  g_Fluid_In     : Global memory array storing the input variables
//                g_Fluid_Out    : Global memory array to store the output variables
//                g_Flux         : Global memory array to store the output fluxes (useful only if StoreFlux == true)
//                dt             : Time interval to advance solution
//                _dh            : 1 / grid size
//                Eta            : Particle mass / Planck constant
//                StoreFlux      : true --> store the coarse-fine fluxes
//                                   --> useful only if CONSERVE_MASS is defined
//                Taylor3_Coeff  : Coefficient in front of the third term in the Taylor expansion
//                j_gap          : Number of useless grids on each side in the j direction (j may not be equal to y)
//                k_gap          : Number of useless grids on each side in the k direction (k mya not be equal to z)
//                s_In           : Shared memory array to store the input data
//                s_Half         : Shared memory array to store the half-step solution
//                s_Flux         : Shared memory array to store the boundary fluxes
//                FinalOut       : true --> store the updated data to g_Fluid_Out
//                XYZ            : 0 : Update the solution in the x direction
//                                 3 : Update the solution in the y direction
//                                 6 : Update the solution in the z direction
//                                 --> This parameter is also used to determine the place to store the output fluxes
//                MinDens        : Minimum allowed density
//-------------------------------------------------------------------------------------------------------
__device__ void CUFLU_Advance( real g_Fluid_In [][FLU_NIN ][ FLU_NXT*FLU_NXT*FLU_NXT ],
                               real g_Fluid_Out[][FLU_NOUT][ PS2*PS2*PS2 ],
                               real g_Flux     [][9][NFLUX_TOTAL][ PS2*PS2 ],
                               const real dt, const real _dh, const real Eta1, const real Eta2, const bool StoreFlux, const real Taylor3_Coeff,
                               const uint j_gap, const uint k_gap, real s_In[][FLU_BLOCK_SIZE_Y][FLU_NXT],
                               real s_Half[][FLU_BLOCK_SIZE_Y][FLU_NXT], real s_Flux1[][PS2+1], real s_Flux2[][PS2+1], const bool FinalOut,
                               const int XYZ, const real MinDens )
{

   const real _Eta1         = (real)1.0/Eta1;
   const real _Eta2         = (real)1.0/Eta2;
   const real dT1           = (real)0.5*dt*_Eta1;
   const real dT2           = (real)0.5*dt*_Eta2;
   const real _Eta12_dh     = (real)0.5*_dh*_Eta1;
   const real _Eta22_dh     = (real)0.5*_dh*_Eta2;
   const real Coeff11       = dT1*_dh*_dh;
   const real Coeff12       = dT2*_dh*_dh;
#  ifdef CONSERVE_MASS
   const real Coeff21       = Taylor3_Coeff*SQR(Coeff11);
   const real Coeff22       = Taylor3_Coeff*SQR(Coeff12);
#  else
   const real Coeff21       = (real)0.5*SQR(Coeff11);
   const real Coeff22       = (real)0.5*SQR(Coeff12);
   const real Coeff31       = Taylor3_Coeff*CUBE(Coeff11);
   const real Coeff32       = Taylor3_Coeff*CUBE(Coeff12);
#  endif

   const uint bx           = blockIdx.x;
   const uint tx           = threadIdx.x;
   const uint ty           = threadIdx.y;
   const uint tid          = __umul24(ty,FLU_BLOCK_SIZE_X) + tx;
   const uint size_j       = FLU_NXT - (j_gap<<1);
   const uint size_k       = FLU_NXT - (k_gap<<1);
   const uint NColumnTotal = __umul24( size_j, size_k );    // total number of data columns to be updated
   const uint i            = tx + FLU_GHOST_SIZE;           // (i,j,k): array indices used in g_Fluid_In
   const uint j_end       = FLU_NXT - j_gap;
         uint j           = j_gap + ty%size_j;
         uint k           = k_gap + ty/size_j;
         uint Column0     = 0;                              // the total number of columns that have been updated
         uint NColumnOnce = MIN( NColumnTotal, FLU_BLOCK_SIZE_Y );

   double Amp1_New, Amp2_New;            // use double precision to reduce the round-off error in the mass conservation
   real   Re1_Old, Im1_Old, Re1_New, Im1_New;
   real   Re2_Old, Im2_Old, Re2_New, Im2_New;
   uint   Idx1, Idx2, Idx3, delta_k;

#  ifdef CONSERVE_MASS
   const uint NThread     = FLU_BLOCK_SIZE_X*FLU_BLOCK_SIZE_Y;
   const uint NHalf       = FLU_NXT - 4*LAP_GHOST;
   const real dT1_dh2     = dT1*_dh*_dh;
   const real dT2_dh2     = dT2*_dh*_dh;
   const uint txp         = tx + 1;

   double Amp1_Old, Amp2_Old, Amp1_Corr, Amp2_Corr;  // use double precision to reduce the round-off error in the mass conservation
   real   R1, I1, dR1, dI1;
   real   R2, I2, dR2, dI2;
   uint   Idx;
   uint   si, sj;                                           // array indices used in the shared memory array
   uint   f, fp1;                                           // array indices used in the s_Flux array
#  ifdef LAPLACIAN_4TH
   uint   fm1, fp2;
#  endif
#  endif // #ifdef CONSERVE_MASS


// determine the array indices for loading the ghost-zone data
   bool LoadGhost = false;                                  // true --> load the ghost-zone data
   uint LoadGhost_i;
   int  LoadGhost_di, LoadGhost_dIdx1;

   if ( tx < 2*FLU_GHOST_SIZE )
   {
      LoadGhost = true;

      if ( tx < FLU_GHOST_SIZE )    LoadGhost_di = -FLU_GHOST_SIZE;
      else                          LoadGhost_di = -FLU_GHOST_SIZE + PS2;

      switch ( XYZ )
      {
         case 0:  LoadGhost_dIdx1 = LoadGhost_di;                                break;
         case 3:  LoadGhost_dIdx1 = __mul24( LoadGhost_di, FLU_NXT );            break;
         case 6:  LoadGhost_dIdx1 = __mul24( LoadGhost_di, FLU_NXT*FLU_NXT );    break;
      }

      LoadGhost_i = (int)i + LoadGhost_di;
   } // if ( tx < 2*FLU_GHOST_SIZE )


// loop over all data columns
   while ( Column0 < NColumnTotal )
   {
//    1. load data into shared memory
      if ( tid < NColumnOnce*PS2 )
      {
//       1.1 determine the array indices for loading global memory data along different directions
         switch ( XYZ )
         {
            case 0:  Idx1 = to1D1( k, j, i );    break;
            case 3:  Idx1 = to1D1( k, i, j );    break;
            case 6:  Idx1 = to1D1( i, k, j );    break;
         }

//       1.2 load the interior data into shared memory
         Re1_Old = g_Fluid_In[bx][0][Idx1];
         Im1_Old = g_Fluid_In[bx][1][Idx1];
         Re2_Old = g_Fluid_In[bx][2][Idx1];
         Im2_Old = g_Fluid_In[bx][3][Idx1];

         s_In[0][ty][i] = Re1_Old;
         s_In[1][ty][i] = Im1_Old;
         s_In[2][ty][i] = Re2_Old;
         s_In[3][ty][i] = Im2_Old;

//       1.3 load the ghost-zone data into shared memory
         if ( LoadGhost )
         {
            s_In[0][ty][LoadGhost_i] = g_Fluid_In[bx][0][ (int)Idx1 + LoadGhost_dIdx1 ];
            s_In[1][ty][LoadGhost_i] = g_Fluid_In[bx][1][ (int)Idx1 + LoadGhost_dIdx1 ];
            s_In[2][ty][LoadGhost_i] = g_Fluid_In[bx][2][ (int)Idx1 + LoadGhost_dIdx1 ];
            s_In[3][ty][LoadGhost_i] = g_Fluid_In[bx][3][ (int)Idx1 + LoadGhost_dIdx1 ];
         }
      } // if ( tid < NColumnOnce*PS2 )

      __syncthreads();


#     ifdef CONSERVE_MASS


//    2. half-step solution
      Idx = tid;
      while ( Idx < NColumnOnce*NHalf )
      {
         si = Idx % NHalf + 2*LAP_GHOST;
         sj = Idx / NHalf;

         s_Half[0][sj][si] = s_In[0][sj][si] - (real)0.5*Coeff11*LAP1( s_In[1][sj], si ) - Coeff21*LAP2( s_In[0][sj], si );
         s_Half[1][sj][si] = s_In[1][sj][si] + (real)0.5*Coeff11*LAP1( s_In[0][sj], si ) - Coeff21*LAP2( s_In[1][sj], si );
         s_Half[2][sj][si] = s_In[2][sj][si] - (real)0.5*Coeff12*LAP1( s_In[3][sj], si ) - Coeff22*LAP2( s_In[2][sj], si );
         s_Half[3][sj][si] = s_In[3][sj][si] + (real)0.5*Coeff12*LAP1( s_In[2][sj], si ) - Coeff22*LAP2( s_In[3][sj], si );

         Idx += NThread;
      } // while ( Idx < NColumnOnce*NHalf )

      __syncthreads();


//    3. calculate the face-center fluxes (the coefficient _dh has been absorted into the constant dT_dh2)
      Idx = tid;
      while ( Idx < NColumnOnce*(PS2+1) )
      {
         si  = Idx % (PS2+1);
         sj  = Idx / (PS2+1);
         f   = si + FLU_GHOST_SIZE - 1;
         fp1 = f + 1;

#        ifdef LAPLACIAN_4TH
         fm1 = f - 1;
         fp2 = f + 2;

         R1  = real(1./28.)*( -s_Half[0][sj][fm1]+(real)15*s_Half[0][sj][f]+(real)15*s_Half[0][sj][fp1]-s_Half[0][sj][fp2] );
         I1  = real(1./28.)*( -s_Half[1][sj][fm1]+(real)15*s_Half[1][sj][f]+(real)15*s_Half[1][sj][fp1]-s_Half[1][sj][fp2] );
         dR1 = real(1./12.)*( +s_Half[0][sj][fm1]-(real)15*s_Half[0][sj][f]+(real)15*s_Half[0][sj][fp1]-s_Half[0][sj][fp2] );
         dI1 = real(1./12.)*( +s_Half[1][sj][fm1]-(real)15*s_Half[1][sj][f]+(real)15*s_Half[1][sj][fp1]-s_Half[1][sj][fp2] );
         R2  = real(1./28.)*( -s_Half[2][sj][fm1]+(real)15*s_Half[2][sj][f]+(real)15*s_Half[2][sj][fp1]-s_Half[2][sj][fp2] );
         I2  = real(1./28.)*( -s_Half[3][sj][fm1]+(real)15*s_Half[3][sj][f]+(real)15*s_Half[3][sj][fp1]-s_Half[3][sj][fp2] );
         dR2 = real(1./12.)*( +s_Half[2][sj][fm1]-(real)15*s_Half[2][sj][f]+(real)15*s_Half[2][sj][fp1]-s_Half[2][sj][fp2] );
         dI2 = real(1./12.)*( +s_Half[3][sj][fm1]-(real)15*s_Half[3][sj][f]+(real)15*s_Half[3][sj][fp1]-s_Half[3][sj][fp2] );

#        else

         R1  = real(0.5)*( + s_Half[0][sj][f] + s_Half[0][sj][fp1] );
         I1  = real(0.5)*( + s_Half[1][sj][f] + s_Half[1][sj][fp1] );
         dR1 =           ( - s_Half[0][sj][f] + s_Half[0][sj][fp1] );
         dI1 =           ( - s_Half[1][sj][f] + s_Half[1][sj][fp1] );
         R2  = real(0.5)*( + s_Half[2][sj][f] + s_Half[2][sj][fp1] );
         I2  = real(0.5)*( + s_Half[3][sj][f] + s_Half[3][sj][fp1] );
         dR2 =           ( - s_Half[2][sj][f] + s_Half[2][sj][fp1] );
         dI2 =           ( - s_Half[3][sj][f] + s_Half[3][sj][fp1] );
#        endif

         s_Flux1[sj][si] = (real)2.0*( R1*dI1 - I1*dR1 );
         s_Flux2[sj][si] = (real)2.0*( R2*dI2 - I2*dR2 );
         Idx += NThread;
      } // while ( Idx < NColumnOnce*(PS2+1) )

      __syncthreads();


//    4a. full-step solution (equivalent to the 3rd-order Taylor expansion)
      if ( tid < NColumnOnce*PS2 )
      {
         Re1_New   = Re1_Old - Coeff11*LAP1( s_Half[1][ty], i );
         Im1_New   = Im1_Old + Coeff11*LAP1( s_Half[0][ty], i );
         Re2_New   = Re2_Old - Coeff12*LAP1( s_Half[3][ty], i );
         Im2_New   = Im2_Old + Coeff12*LAP1( s_Half[2][ty], i );

         Amp1_Old  = SQR( Re1_Old ) + SQR( Im1_Old );
         Amp2_Old  = SQR( Re2_Old ) + SQR( Im2_Old );
         Amp1_New  = SQR( Re1_New ) + SQR( Im1_New );
         Amp2_New  = SQR( Re2_New ) + SQR( Im2_New );
         Amp1_Corr = Amp1_Old - dT1_dh2*( s_Flux1[ty][txp] - s_Flux1[ty][tx] );
         Amp2_Corr = Amp2_Old - dT2_dh2*( s_Flux2[ty][txp] - s_Flux2[ty][tx] );

//       be careful about the negative density and the vacuum (where we might have Amp_New == 0.0)
//       if ( Amp_Corr > (real)0.0  &&  Amp_New > (real)0.0 )
         if ( Amp1_Corr >       0.0  &&  Amp1_New >       0.0 )
         {
            /*
            Re_New *= SQRT( Amp_Corr / Amp_New );
            Im_New *= SQRT( Amp_Corr / Amp_New );
            */
            Re1_New *= sqrt( Amp1_Corr / Amp1_New );  // use double precision to improve the mass conservation further
            Im1_New *= sqrt( Amp1_Corr / Amp1_New );
            Amp1_New = Amp1_Corr;
         }
         if ( Amp2_Corr >       0.0  &&  Amp2_New >       0.0 )
         {
            Re2_New *= sqrt( Amp2_Corr / Amp2_New );
            Im2_New *= sqrt( Amp2_Corr / Amp2_New );
            Amp2_New = Amp2_Corr;
         }
      } // if if ( tid < NColumnOnce*PS2 )


#     else // CONSERVE_MASS


//    4b. full-step solution if CONSERVE_MASS is not defined (equivalent to the 3rd-order Taylor expansion)
      if ( tid < NColumnOnce*PS2 )
      {
         Re1_New  = Re1_Old - Coeff11*LAP1( s_In[1][ty], i ) - Coeff21*LAP2( s_In[0][ty], i ) + Coeff31*LAP3( s_In[1][ty], i );
         Im1_New  = Im1_Old + Coeff11*LAP1( s_In[0][ty], i ) - Coeff21*LAP2( s_In[1][ty], i ) - Coeff31*LAP3( s_In[0][ty], i );
         Re2_New  = Re2_Old - Coeff12*LAP1( s_In[3][ty], i ) - Coeff22*LAP2( s_In[2][ty], i ) + Coeff32*LAP3( s_In[3][ty], i );
         Im2_New  = Im2_Old + Coeff12*LAP1( s_In[2][ty], i ) - Coeff22*LAP2( s_In[3][ty], i ) - Coeff32*LAP3( s_In[2][ty], i );
         Amp1_New = SQR( Re1_New ) + SQR( Im1_New );
         Amp2_New = SQR( Re2_New ) + SQR( Im2_New );
      }


#     endif // CONSERVE_MASS ... else ...


//    5. store the updated data (and fluxes) back to the global memory
      if ( tid < NColumnOnce*PS2 )
      {
//       5.1 data
         if ( FinalOut )
         {
//          apply the the minimum density check
//          --> to be consistent with the CPU solver, we apply it just before storing the output results to g_Fluid_Out
            if ( Amp1_New < MinDens )
            {
               const real Rescale1 = SQRT( MinDens / (real)Amp1_New );

               Re1_New *= Rescale1;
               Im1_New *= Rescale1;
               Amp1_New = MinDens;
            }
            if ( Amp2_New < MinDens )
            {
               const real Rescale2 = SQRT( MinDens / (real)Amp2_New );

               Re2_New *= Rescale2;
               Im2_New *= Rescale2;
               Amp2_New = MinDens;
            }

            switch ( XYZ )
            {
               case 0:  Idx2 = to1D2( k, j, i );    break;
               case 3:  Idx2 = to1D2( k, i, j );    break;
               case 6:  Idx2 = to1D2( i, k, j );    break;
            }

            g_Fluid_Out[bx][0][Idx2] = Amp1_New;
            g_Fluid_Out[bx][1][Idx2] = Re1_New;
            g_Fluid_Out[bx][2][Idx2] = Im1_New;
            g_Fluid_Out[bx][3][Idx2] = Amp2_New;
            g_Fluid_Out[bx][4][Idx2] = Re2_New;
            g_Fluid_Out[bx][5][Idx2] = Im2_New;
         }

         else
         {
            g_Fluid_In[bx][0][Idx1] = Re1_New;
            g_Fluid_In[bx][1][Idx1] = Im1_New;
            g_Fluid_In[bx][2][Idx1] = Re2_New;
            g_Fluid_In[bx][3][Idx1] = Im2_New;
         }


//       5.2 fluxes (for the flux-correction operation)
         if ( StoreFlux  &&  tx == 0 )
         if ( k >= FLU_GHOST_SIZE  &&  k < FLU_NXT-FLU_GHOST_SIZE )
         if ( j >= FLU_GHOST_SIZE  &&  j < FLU_NXT-FLU_GHOST_SIZE )
         {
            Idx3 = __umul24( k-FLU_GHOST_SIZE, PS2 ) + (j-FLU_GHOST_SIZE);

            g_Flux[bx][XYZ+0][0][Idx3] = s_Flux1[ty][  0]*_Eta12_dh;
            g_Flux[bx][XYZ+1][0][Idx3] = s_Flux1[ty][PS1]*_Eta12_dh;
            g_Flux[bx][XYZ+2][0][Idx3] = s_Flux1[ty][PS2]*_Eta12_dh;
            g_Flux[bx][XYZ+0][1][Idx3] = s_Flux2[ty][  0]*_Eta22_dh;
            g_Flux[bx][XYZ+1][1][Idx3] = s_Flux2[ty][PS1]*_Eta22_dh;
            g_Flux[bx][XYZ+2][1][Idx3] = s_Flux2[ty][PS2]*_Eta22_dh;
         }


//       5.3 reset the target array indices
         j += NColumnOnce;

         if ( j >= j_end )
         {
            delta_k  = ( j - j_end )/size_j + 1;
            k       += delta_k;
            j       -= __umul24( size_j, delta_k );
         }
      } // if ( tid < NColumnOnce*PS2 )

      __syncthreads();

      Column0     += NColumnOnce;
      NColumnOnce  = MIN( NColumnTotal - Column0, FLU_BLOCK_SIZE_Y );

   } // while ( Column0 < NColumnTotal )

} // FUNCTION : CUFLU_Advance



#endif // #if ( defined GPU  &&  MODEL == ELBDM )
