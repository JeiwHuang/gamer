#include "hip/hip_runtime.h"
#include "Macro.h"
#include "CUPOT.h"

#if ( defined GPU  &&  MODEL == ELBDM  &&  defined GRAVITY )



#include "../../SelfGravity/GPU_Poisson/CUPOT_ExternalPot.cu"

// variables reside in constant memory
__constant__ double ExtPot_AuxArray_d[EXT_POT_NAUX_MAX];




//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_ELBDMGravitySolver_SetConstMem
// Description :  Set the constant memory used by CUPOT_ELBDMGravitySolver
//
// Note        :  Adopt the suggested approach for CUDA version >= 5.0
//
// Parameter   :  None
//
// Return      :  0/-1 : successful/failed
//---------------------------------------------------------------------------------------------------
int CUPOT_ELBDMGravitySolver_SetConstMem( double ExtPot_AuxArray_h[] )
{

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( ExtPot_AuxArray_d), ExtPot_AuxArray_h, EXT_POT_NAUX_MAX*sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -1;

   else
      return 0;

} // FUNCTION : CUPOT_ELBDMGravitySolver_SetConstMem



//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_ELBDMGravitySolver
// Description :  GPU ELBDM gravity solver
//                --> Use GPU to advance wave function by exp( -i*Eta*(Phi+Lambda*Rho)*dt )
//
// Note        :  1. ELBDM gravity solver requires NO potential and fluid ghost zone
//                   --> Optimized performance can be achieved if GRA_GHOST_SIZE == 0, GRA_NXT == PATCH_SIZE
//                   --> But the code supports GRA_GHOST_SIZE > 0 as well (mainly for the STORE_POT_GHOST option) 
//                2. ELBDM gravity solver does NOT need the density information (if QUARTIC_SELF_INTERACTION is off)
//                   --> DENS component will NOT be sent in and out in this solver
//                   --> GRA_NIN == 2 (only store the real and imaginary parts)
//                   --> If QUARTIC_SELF_INTERACTION is on, the density is *calculated* here to be REAL^2+IMAG^2
//                3. Prefix "g" for pointers pointing to the "Global" memory space
//                   Prefix "s" for pointers pointing to the "Shared" memory space
//                4. No shared memory is used in this kernel since no computational stencil is required
//                   and hence no data needed to be shared
//
// Parameter   :  g_Flu_Array    : Global memory array to store the input and output data
//                g_Pot_Array    : Global memory array storing the input potential for evaluating the
//                                 gravitational acceleration
//                g_Corner_Array : Global memory array storing the physical corner coordinates of each patch
//                EtaDt          : Particle mass / Planck constant * dt
//                dh             : Cell size
//                Lambda         : Quartic self-interaction coefficient in ELBDM
//                ExtPot         : Add the external potential
//                Time           : Physical time (may be used by CUPOT_ExternalPot)
//---------------------------------------------------------------------------------------------------
__global__ void CUPOT_ELBDMGravitySolver(       real g_Flu_Array[][GRA_NIN][ PS1*PS1*PS1 ],
                                          const real g_Pot_Array[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                          const double g_Corner_Array[][3],
                                          const real EtaDt, const real dh, const real Lambda, const bool ExtPot,
                                          const double Time )
{

   const uint bx      = blockIdx.x;
   const uint tx      = threadIdx.x; 
   const uint ty      = threadIdx.y; 
   const uint tz      = threadIdx.z; 
         uint Idx_Flu =  __umul24( tz, PS1*PS1 ) + __umul24( ty, PS1 ) + tx;
         uint Idx_Pot =  __umul24( tz+GRA_GHOST_SIZE, GRA_NXT*GRA_NXT )
                       + __umul24( ty+GRA_GHOST_SIZE, GRA_NXT )
                       +           tx+GRA_GHOST_SIZE;

   real   Re, Im, Phase, Cos_Phase, Sin_Phase, Pot;
   double x, y, z;


   if ( ExtPot )
   {
      x = g_Corner_Array[bx][0] + (double)(tx*dh);
      y = g_Corner_Array[bx][1] + (double)(ty*dh);
   }

   for (uint k=tz; k<PS1; k+=GRA_BLOCK_SIZE_Z)
   {
      Re        = g_Flu_Array[bx][0][Idx_Flu];
      Im        = g_Flu_Array[bx][1][Idx_Flu];
      Pot       = g_Pot_Array[bx]   [Idx_Pot];

#     ifdef QUARTIC_SELF_INTERACTION
      Pot      += Lambda*( SQR(Re) + SQR(Im) );
#     endif

      if ( ExtPot ) {
      z         = g_Corner_Array[bx][2] + (double)(k*dh);
      Pot      += CUPOT_ExternalPot( x, y, z, Time, ExtPot_AuxArray_d ); }

      Phase     = EtaDt * Pot;
      Cos_Phase = COS( Phase );
      Sin_Phase = SIN( Phase );

      g_Flu_Array[bx][0][Idx_Flu] = Cos_Phase*Re + Sin_Phase*Im;
      g_Flu_Array[bx][1][Idx_Flu] = Cos_Phase*Im - Sin_Phase*Re;

      Idx_Flu += GRA_BLOCK_SIZE_Z*PS1*PS1;
      Idx_Pot += GRA_BLOCK_SIZE_Z*GRA_NXT*GRA_NXT;

   } // for (uint k=tz; k<PS1; k+=GRA_BLOCK_SIZE_Z)

} // FUNCTION : CUPOT_ELBDMGravitySolver



#endif // #if ( defined GPU  &&  MODEL == ELBDM  &&  defined GRAVITY )
