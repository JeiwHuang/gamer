#include "hip/hip_runtime.h"
#include "Macro.h"
#include "CUPOT.h"

#if ( defined GPU  &&  MODEL == ELBDM  &&  defined GRAVITY )



#include "../../SelfGravity/GPU_Poisson/CUPOT_ExternalPot.cu"

// variables reside in constant memory
__constant__ double ExtPot_AuxArray_d[EXT_POT_NAUX_MAX];




//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_ELBDMGravitySolver_SetConstMem
// Description :  Set the constant memory used by CUPOT_ELBDMGravitySolver
//
// Note        :  Adopt the suggested approach for CUDA version >= 5.0
//
// Parameter   :  None
//
// Return      :  0/-1 : successful/failed
//---------------------------------------------------------------------------------------------------
int CUPOT_ELBDMGravitySolver_SetConstMem( double ExtPot_AuxArray_h[] )
{

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( ExtPot_AuxArray_d), ExtPot_AuxArray_h, EXT_POT_NAUX_MAX*sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -1;

   else
      return 0;

} // FUNCTION : CUPOT_ELBDMGravitySolver_SetConstMem



//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_ELBDMGravitySolver
// Description :  GPU ELBDM gravity solver
//                --> Use GPU to advance wave function by exp( -i*Eta*(Phi+Lambda*Rho)*dt )
//
// Note        :  1. ELBDM gravity solver requires NO potential and fluid ghost zone
//                   --> Optimized performance can be achieved if GRA_GHOST_SIZE == 0, GRA_NXT == PATCH_SIZE
//                   --> But the code supports GRA_GHOST_SIZE > 0 as well (mainly for the STORE_POT_GHOST option) 
//                2. ELBDM gravity solver does NOT need the density information (if QUARTIC_SELF_INTERACTION is off)
//                   --> DENS component will NOT be sent in and out in this solver
//                   --> GRA_NIN == 2 (only store the real and imaginary parts)
//                   --> If QUARTIC_SELF_INTERACTION is on, the density is *calculated* here to be REAL^2+IMAG^2
//                3. Prefix "g" for pointers pointing to the "Global" memory space
//                   Prefix "s" for pointers pointing to the "Shared" memory space
//                4. No shared memory is used in this kernel since no computational stencil is required
//                   and hence no data needed to be shared
//
// Parameter   :  g_Flu_Array    : Global memory array to store the input and output data
//                g_Pot_Array    : Global memory array storing the input potential for evaluating the
//                                 gravitational acceleration
//                g_Corner_Array : Global memory array storing the physical corner coordinates of each patch
//                EtaDt          : Particle mass / Planck constant * dt
//                dh             : Cell size
//                Lambda         : Quartic self-interaction coefficient in ELBDM
//                ExtPot         : Add the external potential
//                Time           : Physical time (may be used by CUPOT_ExternalPot)
//---------------------------------------------------------------------------------------------------
__global__ void CUPOT_ELBDMGravitySolver(       real g_Flu_Array[][GRA_NIN][ PS1*PS1*PS1 ],
                                          const real g_Pot_Array[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                          const double g_Corner_Array[][3],
                                          const real Eta1Dt, const real Eta2Dt, const real dh, const real Lambda, const bool ExtPot,
                                          const double Time )
{

   const uint bx      = blockIdx.x;
   const uint tx      = threadIdx.x; 
   const uint ty      = threadIdx.y; 
   const uint tz      = threadIdx.z; 
         uint Idx_Flu =  __umul24( tz, PS1*PS1 ) + __umul24( ty, PS1 ) + tx;
         uint Idx_Pot =  __umul24( tz+GRA_GHOST_SIZE, GRA_NXT*GRA_NXT )
                       + __umul24( ty+GRA_GHOST_SIZE, GRA_NXT )
                       +           tx+GRA_GHOST_SIZE;

   real   Re1, Im1, Re2, Im2, Phase1, Phase2, Cos_Phase1, Sin_Phase1, Cos_Phase2, Sin_Phase2, Pot;
   double x, y, z;


   if ( ExtPot )
   {
      x = g_Corner_Array[bx][0] + (double)(tx*dh);
      y = g_Corner_Array[bx][1] + (double)(ty*dh);
   }

   for (uint k=tz; k<PS1; k+=GRA_BLOCK_SIZE_Z)
   {
      Re1        = g_Flu_Array[bx][0][Idx_Flu];
      Im1        = g_Flu_Array[bx][1][Idx_Flu];
      Re2        = g_Flu_Array[bx][2][Idx_Flu];
      Im2        = g_Flu_Array[bx][3][Idx_Flu];
      Pot       = g_Pot_Array[bx]   [Idx_Pot];

#     ifdef QUARTIC_SELF_INTERACTION
      Pot      += Lambda*( SQR(Re1) + SQR(Im1) + SQR(Re2) + SQR(Im2) );
#     endif

      if ( ExtPot ) {
      z         = g_Corner_Array[bx][2] + (double)(k*dh);
      Pot      += CUPOT_ExternalPot( x, y, z, Time, ExtPot_AuxArray_d ); }

      Phase1     = Eta1Dt * Pot;
      Phase2     = Eta2Dt * Pot;
      Cos_Phase1 = COS( Phase1 );
      Sin_Phase1 = SIN( Phase1 );
      Cos_Phase2 = COS( Phase2 );
      Sin_Phase2 = SIN( Phase2 );

      g_Flu_Array[bx][0][Idx_Flu] = Cos_Phase1*Re1 + Sin_Phase1*Im1;
      g_Flu_Array[bx][1][Idx_Flu] = Cos_Phase1*Im1 - Sin_Phase1*Re1;
      g_Flu_Array[bx][2][Idx_Flu] = Cos_Phase2*Re2 + Sin_Phase2*Im2;
      g_Flu_Array[bx][3][Idx_Flu] = Cos_Phase2*Im2 - Sin_Phase2*Re2;

      Idx_Flu += GRA_BLOCK_SIZE_Z*PS1*PS1;
      Idx_Pot += GRA_BLOCK_SIZE_Z*GRA_NXT*GRA_NXT;

   } // for (uint k=tz; k<PS1; k+=GRA_BLOCK_SIZE_Z)

} // FUNCTION : CUPOT_ELBDMGravitySolver



#endif // #if ( defined GPU  &&  MODEL == ELBDM  &&  defined GRAVITY )
